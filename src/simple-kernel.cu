#include "hip/hip_runtime.h"
#if defined(__HIP)
#include "hip/hip_runtime.h"
#endif

#include "utils.h"

/* CUDA Kernel to multipy a vector with 1.25 for a number of iterations.*/
__global__ void vectorIterMult(DataType* v, unsigned int v_len,
                               unsigned int iter)
{
  const int idx = threadIdx.x + blockDim.x * blockIdx.x;

  if (idx >= v_len)
    return;

  for (unsigned int i = 0; i < iter; ++i) {
    v[idx] = 1.00005 * v[idx];
  }
}
