#include "hip/hip_runtime.h"
#include "device.h"
#include "host.h"
#include "utils.h"
#include <stdio.h>
#include <stdlib.h>

#if defined(__HIP)
#include "hip/hip_runtime.h"
#define ACC(x) hip##x
#else
#define ACC(x) cuda##x
#endif

#define printCudaError(cuda_returned_error_code)                               \
  {                                                                            \
    accErrorPrint((cuda_returned_error_code), __FILE__, __LINE__);             \
  }

inline void accErrorPrint(ACC(Error_t) code, const char* file, int line)
{
  fprintf(stderr, "ACC Error: %s (%d) %s %d\n", ACC(GetErrorString)(code), code,
          file, line);
}

static DataType euclicianNormVector(DataType* vectorA, unsigned int length)
{
  DataType diffEu = 0;

  for (int i = 0; i < length; ++i) {
    diffEu += vectorA[i] * vectorA[i];
  }
  diffEu = sqrt(diffEu);

  return diffEu;
}

#include "simple-kernel.cu"

static DataType* gd_vector;
static DataType* device_setup(DataType* h_vector, unsigned int v_len)
{
  ACC(Error_t) device_error;
  device_error = ACC(Malloc)(&gd_vector, sizeof(DataType) * v_len);
  if (device_error != ACC(Success)) {
    printCudaError(device_error);
    return NULL;
  }

  device_error = ACC(Memcpy)(gd_vector, h_vector, sizeof(DataType) * v_len,
                             ACC(MemcpyHostToDevice));
  if (device_error != ACC(Success)) {
    printCudaError(device_error);
    return NULL;
  }

  return gd_vector;
}

void device_teardown(void) { ACC(Free)(gd_vector); }

enum time_categories {
  TOTAL_TIME,
  CUDA_DIFF_TIME,
  GRAPH_CREATION,
  TOTAL_LAUNCH_COST,
  EXEC_TIME,
};
#define TOTAL_NR_TIMES (1 + EXEC_TIME - TOTAL_TIME)
static double time_elapsed[TOTAL_NR_TIMES];
static void reset_times(void)
{
  for (int i = 0; i < TOTAL_NR_TIMES; ++i) {
    time_elapsed[i] = 0.0;
  }
}

#define TPB 1024
int device_kernel_run(const struct options* opt, DataType* d_vector)
{
  dim3 block(TPB);
  dim3 grid((opt->number_of_threads + TPB - 1) / TPB);

#ifdef TIME_DETAILED
  double time_start;
  time_start = getCpuSeconds();
#endif
  for (unsigned int i = 0; i < opt->outer_iterations; ++i) {
    for (unsigned int k = 0; k < opt->number_of_kernels; ++k) {
      vectorIterMult<<<grid, block>>>(d_vector, opt->number_of_threads,
                                      opt->inner_iterations);
    }
  }
#ifdef TIME_DETAILED
  time_elapsed[TOTAL_LAUNCH_COST] = getCpuSeconds() - time_start;
#endif
  ACC(DeviceSynchronize)();
#ifdef TIME_DETAILED
  time_elapsed[EXEC_TIME] = getCpuSeconds() - time_start;
#endif
  return 0;
}

static ACC(Graph_t) g_main_graph;
static ACC(GraphNode_t) * g_nodes;
static ACC(GraphExec_t) g_exec_work_graph;
static ACC(Stream_t) g_stream_for_cuda_graph;

ACC(Error_t) device_graph_setup(const struct options* opt, DataType** d_vector)
{
#ifdef TIME_DETAILED
  double time_start = getCpuSeconds();
#endif
  ACC(Error_t) device_error;
  device_error = ACC(GraphCreate)(&g_main_graph, 0);
  if (ACC(Success) != device_error) {
    printCudaError(device_error);
    return device_error;
  }

  dim3 block(TPB);
  dim3 grid((opt->number_of_threads + TPB - 1) / TPB);

  void* ka_kernel[] = {(void*) d_vector, (void*) &opt->number_of_threads,
                       (void*) &opt->inner_iterations};
  ACC(KernelNodeParams) np_kernel = {0};
  np_kernel.func = (void*) vectorIterMult;
  np_kernel.gridDim = grid;
  np_kernel.blockDim = block;
  np_kernel.kernelParams = ka_kernel;

  ACC(GraphNode_t)* last_node = NULL;
  unsigned int num_dependencies = 0;
  g_nodes = (ACC(GraphNode_t)*) malloc(opt->number_of_kernels *
                                       sizeof(ACC(GraphNode_t)));
  for (unsigned int i = 0; i < opt->number_of_kernels; ++i) {
    device_error = ACC(GraphAddKernelNode)(&g_nodes[i], g_main_graph, last_node,
                                           num_dependencies, &np_kernel);

    if (ACC(Success) != device_error) {
      printCudaError(device_error);
      return device_error;
    }
    last_node = &g_nodes[i];
    num_dependencies = 1;
  }

  device_error =
      ACC(GraphInstantiateWithFlags)(&g_exec_work_graph, g_main_graph, 0);
  if (ACC(Success) != device_error) {
    printCudaError(device_error);
    return device_error;
  }

  device_error = ACC(StreamCreateWithFlags)(&g_stream_for_cuda_graph,
                                            ACC(StreamNonBlocking));
  if (ACC(Success) != device_error) {
    printCudaError(device_error);
    return device_error;
  }
  device_error = ACC(GraphUpload)(g_exec_work_graph, g_stream_for_cuda_graph);
  if (ACC(Success) != device_error) {
    printCudaError(device_error);
    return device_error;
  }
#ifdef TIME_DETAILED
  time_elapsed[GRAPH_CREATION] = getCpuSeconds() - time_start;
#endif
  return ACC(Success);
}

ACC(Error_t) device_graph_run(const struct options* opt)
{
#ifdef TIME_DETAILED
  double time_start;
  time_start = getCpuSeconds();
#endif
  for (unsigned int i = 0; i < opt->outer_iterations; ++i) {
    ACC(GraphLaunch)(g_exec_work_graph, g_stream_for_cuda_graph);
  }
#ifdef TIME_DETAILED
  time_elapsed[TOTAL_LAUNCH_COST] = getCpuSeconds() - time_start;
#endif
  ACC(StreamSynchronize)(g_stream_for_cuda_graph);
#ifdef TIME_DETAILED
  time_elapsed[EXEC_TIME] = getCpuSeconds() - time_start;
#endif
#ifdef MEM_CHECK
#if defined(__HIP)
  system("rocm-smi --showmeminfo vram");
#else
  system("nvidia-smi --query-gpu=memory.used --format=csv --id=0");
#endif
#endif
  return ACC(Success);
}

void device_graph_teardown(void)
{
  ACC(StreamDestroy)(g_stream_for_cuda_graph);
  ACC(GraphExecDestroy)(g_exec_work_graph);
  ACC(GraphDestroy)(g_main_graph);
  free(g_nodes);
}

int device_run(const struct options* opt)
{
  ACC(Error_t) device_error;
  double time_start[2] = {0, 0};
  reset_times();
  time_start[0] = getCpuSeconds();

  /* Generate host data. */
  DataType* h_vector;
  h_vector = host_setup(opt->number_of_threads, opt->seed);
  if (NULL == h_vector) {
    return -1;
  }

  DataType* d_vector;
  if (opt->run_cpu) {
    cpu_kernel_run(opt, h_vector);
  } else {
    /* Setup device resources. */
    d_vector = device_setup(h_vector, opt->number_of_threads);
    if (NULL == d_vector) {
      device_teardown();
      host_teardown();
      return -1;
    }

    time_start[1] = getCpuSeconds();
    if (opt->run_graph) {
      device_error = device_graph_setup(opt, &d_vector);
      if (ACC(Success) == device_error) {
        device_graph_run(opt);
      }
    } else {
      device_kernel_run(opt, d_vector);
    }
    time_elapsed[CUDA_DIFF_TIME] = getCpuSeconds() - time_start[1];

    device_error = ACC(Memcpy)(h_vector, d_vector,
                               sizeof(DataType) * opt->number_of_threads,
                               ACC(MemcpyDeviceToHost));
    if (device_error != ACC(Success)) {
      printCudaError(device_error);
    }

    /* teardown device resources. */
    device_teardown();
    if (opt->run_graph) {
      device_graph_teardown();
    }
  }

  DataType result = euclicianNormVector(h_vector, opt->number_of_threads);

  /* teardown host resources. */
  host_teardown();

  time_elapsed[TOTAL_TIME] = getCpuSeconds() - time_start[0];

  print_times(opt, time_elapsed, TOTAL_NR_TIMES, result);
  return 0;
}
